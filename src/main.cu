#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <stdio.h>
#include <sys/time.h>

using namespace std;

constexpr int b_ = 1024;
constexpr int m_ = 1024;
constexpr int n_ = 1024;
constexpr int k_ = 1024;
constexpr int n_iter = 50;

int cublas_test(float* a, float* b, float* c) {
    float* d_a, *d_b, *d_c;
    size_t size_a = sizeof(float) * b_ * m_ * k_;
    size_t size_b = sizeof(float) * b_ * n_ * k_;
    size_t size_c = sizeof(float) * b_ * m_ * n_;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size_c, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasStatus_t ret;
    ret = hipblasCreate(&handle);

    float *a_array[b_], *b_array[b_];
    float *c_array[b_];
    for (int i = 0; i < b_; ++i) {
        a_array[i] = d_a + i * m_ * k_;
        b_array[i] = d_b + i * k_ * n_;
        c_array[i] = d_c + i * m_ * n_;
    }
    const float **d_Aarray, **d_Barray;
    float **d_Carray;
    hipMalloc((void**)&d_Aarray, b_*sizeof(float *));
    hipMalloc((void**)&d_Barray, b_*sizeof(float *));
    hipMalloc((void**)&d_Carray, b_*sizeof(float *));
    hipMemcpy(d_Aarray, a_array, b_*sizeof(float *), hipMemcpyHostToDevice);
    hipMemcpy(d_Barray, b_array, b_*sizeof(float *), hipMemcpyHostToDevice);
    hipMemcpy(d_Carray, c_array, b_*sizeof(float *), hipMemcpyHostToDevice);

    const float alpha  =  1.0f;
    const float beta  =  0.0f;
    int m = m_;
    int n = n_;
    int k = k_;
    int lda = m_;
    int ldb = k_;
    int ldc = m_;
    int batch = b_;

    cout << "start hipblasSgemmBatched benchmark" << endl;

    struct timeval start, end;
    gettimeofday(&start, NULL);
    for (int i = 0; i < n_iter; i++) {
        ret = hipblasSgemmBatched(handle,
                            HIPBLAS_OP_N,
                            HIPBLAS_OP_N,
                            m,n,k,
                            &alpha,
                            d_Aarray,  lda,
                            d_Barray,  ldb,
                            &beta,
                            d_Carray,  ldc,
                            batch);
    }
    gettimeofday(&end, NULL);

    // if (ret == CUBLAS_STATUS_SUCCESS) {
    //     printf("sgemm success  %d, line(%d)\n", ret, __LINE__);
    // }
    printf("[INFO] batch_size %d m %d n %d k %d cublasSgemmBatched-time %.5f ms ( %d iterations) \n", b_, m_, n_, k_,
          ((end.tv_sec - start.tv_sec) * 1000 + (end.tv_usec - start.tv_usec) * 0.001) / n_iter, n_iter);

    hipblasDestroy(handle);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_Aarray);
    hipFree(d_Barray);
    hipFree(d_Carray);
    return 0;
};

int main() {

    float* a = new float[b_ * m_ * k_];
    for(int i = 0; i < b_ * m_ * k_; i++) a[i] = i;

    float* b = new float[b_ * k_ * n_];
    for(int i = 0; i < b_ * k_ * n_; i++) b[i] = i+1;

    float* c = new float[b_ * m_ * n_];
    for(int i = 0; i < b_ * m_ * n_; i++) c[i] = 0.0;

    cublas_test(a, b, c);

    return 0;
}